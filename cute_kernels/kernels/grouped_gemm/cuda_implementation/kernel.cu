// **************************************************
// Copyright (c) 2025, Mayank Mishra
// **************************************************

#include <torch/extension.h>

#include <fstream>
#include <iostream>
#include <sstream>
#include <vector>

#include "cute/tensor.hpp"
#include "cutlass/cutlass.h"
#include "cutlass/epilogue/collective/collective_builder.hpp"
#include "cutlass/epilogue/collective/default_epilogue.hpp"
#include "cutlass/epilogue/thread/linear_combination.h"
#include "cutlass/gemm/collective/collective_builder.hpp"
#include "cutlass/gemm/device/gemm_universal_adapter.h"
#include "cutlass/gemm/dispatch_policy.hpp"
#include "cutlass/gemm/group_array_problem_shape.hpp"
#include "cutlass/gemm/kernel/gemm_universal.hpp"
#include "cutlass/tensor_ref.h"
#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/packed_stride.hpp"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/tensor_view_io.h"
#include "include/cute_kernels.h"

namespace ck = cute_kernels;

using uint32 = ck::uint32;
using int64 = ck::int64;

struct GpuTimer {
    hipStream_t _stream_id;
    hipEvent_t _start;
    hipEvent_t _stop;

    /// Constructor
    GpuTimer() : _stream_id(0) {
        hipEventCreate(&_start);
        hipEventCreate(&_stop);
    }

    /// Destructor
    ~GpuTimer() {
        hipEventDestroy(_start);
        hipEventDestroy(_stop);
    }

    /// Start the timer for a given stream (defaults to the default stream)
    void start(hipStream_t stream_id = 0) {
        _stream_id = stream_id;
        hipEventRecord(_start, _stream_id);
    }

    /// Stop the timer
    void stop() { hipEventRecord(_stop, _stream_id); }

    /// Return the elapsed time (in milliseconds)
    float elapsed_millis() {
        float elapsed = 0.0;
        hipEventSynchronize(_stop);
        hipEventElapsedTime(&elapsed, _start, _stop);
        return elapsed;
    }
};

using namespace cute;

using ProblemShape = cutlass::gemm::GroupProblemShape<Shape<int, int, int>>;  // <M,N,K> per group

using ElementA = cutlass::bfloat16_t;
using ElementB = cutlass::bfloat16_t;
using ElementC = cutlass::bfloat16_t;
using ElementAccumulator = float;

using LayoutA = cutlass::layout::RowMajor;
using LayoutB = cutlass::layout::ColumnMajor;
using LayoutC = cutlass::layout::ColumnMajor;

constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;
constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;
constexpr int AlignmentC = 128 / cutlass::sizeof_bits<ElementC>::value;

using ArchTag = cutlass::arch::Sm100;
using OperatorClass = cutlass::arch::OpClassTensorOp;
using StageCountType = cutlass::gemm::collective::StageCountAuto;

using ClusterShape = Shape<int32_t, int32_t, _1>;

using MmaTileShape = Shape<_256, _256, Int<128 / sizeof(ElementA)>>;

using KernelSchedule = cutlass::gemm::KernelPtrArrayTmaWarpSpecialized2SmSm100;
using EpilogueSchedule = cutlass::epilogue::PtrArrayTmaWarpSpecialized2Sm;

using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
    ArchTag,
    OperatorClass,
    MmaTileShape,
    ClusterShape,
    cutlass::epilogue::collective::EpilogueTileAuto,
    ElementAccumulator,
    ElementAccumulator,
    ElementC,
    LayoutC *,
    AlignmentC,
    ElementC,
    LayoutC *,
    AlignmentC,
    EpilogueSchedule,
    cutlass::epilogue::fusion::LinearCombination<ElementC, ElementAccumulator>>::CollectiveOp;

using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
    ArchTag,
    OperatorClass,
    ElementA,
    LayoutA *,
    AlignmentA,
    ElementB,
    LayoutB *,
    AlignmentB,
    ElementAccumulator,
    MmaTileShape,
    ClusterShape,
    cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
        sizeof(typename CollectiveEpilogue::SharedStorage))>,
    KernelSchedule>::CollectiveOp;

using GemmKernel = cutlass::gemm::kernel::GemmUniversal<ProblemShape, CollectiveMainloop, CollectiveEpilogue>;
using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

// Reference device GEMM implementation type
using DeviceGemmReference = cutlass::reference::device::
    Gemm<ElementA, LayoutA, ElementB, LayoutB, ElementC, LayoutC, ElementAccumulator, ElementAccumulator>;

using StrideA = typename Gemm::GemmKernel::InternalStrideA;
using StrideB = typename Gemm::GemmKernel::InternalStrideB;
using StrideC = typename Gemm::GemmKernel::InternalStrideC;
using StrideD = typename Gemm::GemmKernel::InternalStrideD;

// Host-side allocations
std::vector<int64_t> offset_A;
std::vector<int64_t> offset_B;
std::vector<int64_t> offset_C;

std::vector<StrideA> stride_A_host;
std::vector<StrideB> stride_B_host;
std::vector<StrideC> stride_C_host;

// Device-side allocations
cutlass::DeviceAllocation<typename ProblemShape::UnderlyingProblemShape> problem_sizes;

cutlass::DeviceAllocation<typename Gemm::ElementA> block_A;
cutlass::DeviceAllocation<typename Gemm::ElementB> block_B;
cutlass::DeviceAllocation<typename Gemm::ElementC> block_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput> block_ref_D;

cutlass::DeviceAllocation<const typename Gemm::ElementA *> ptr_A;
cutlass::DeviceAllocation<const typename Gemm::ElementB *> ptr_B;
cutlass::DeviceAllocation<const typename Gemm::ElementC *> ptr_C;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput *> ptr_D;
cutlass::DeviceAllocation<typename Gemm::EpilogueOutputOp::ElementOutput *> ptr_ref_D;

cutlass::DeviceAllocation<StrideA> stride_A;
cutlass::DeviceAllocation<StrideB> stride_B;
cutlass::DeviceAllocation<StrideC> stride_C;

using RasterOrderOptions = typename cutlass::gemm::kernel::detail::PersistentTileSchedulerSm100GroupParams<
    typename ProblemShape::UnderlyingProblemShape>::RasterOrderOptions;

/// Compute performance in GFLOP/s
double get_gflops(const double &runtime_s,
                  std::vector<typename ProblemShape::UnderlyingProblemShape> &problem_sizes_host) {
    // Number of real-valued multiply-adds
    uint64_t fmas = 0;

    for (auto const &problem : problem_sizes_host) {
        fmas += static_cast<uint64_t>(get<0>(problem)) * static_cast<uint64_t>(get<1>(problem)) *
                static_cast<uint64_t>(get<2>(problem));
    }
    // Two flops per multiply-add
    uint64_t flop = uint64_t(2) * uint64_t(fmas);
    double gflop = double(flop) / double(1.0e9);
    return gflop / runtime_s;
}

/// Helper to initialize a block of device data
template <class Element>
void initialize_block(cutlass::DeviceAllocation<Element> &block, uint64_t seed = 2023) {
    cutlass::reference::device::BlockFillRandomUniform(
        block.get(), block.size(), seed, static_cast<Element>(8), static_cast<Element>(-8), 0);
}

/// Allocates device-side data
void allocate(const std::vector<typename ProblemShape::UnderlyingProblemShape> &problem_sizes_host) {
    int64_t total_elements_A = 0;
    int64_t total_elements_B = 0;
    int64_t total_elements_C = 0;

    const uint32 E = problem_sizes_host.size();

    for (uint32 i = 0; i < E; i++) {
        auto problem = problem_sizes_host.at(i);
        auto M = get<0>(problem);
        auto N = get<1>(problem);
        auto K = get<2>(problem);

        offset_A.push_back(total_elements_A);
        offset_B.push_back(total_elements_B);
        offset_C.push_back(total_elements_C);

        int64_t elements_A = M * K;
        int64_t elements_B = K * N;
        int64_t elements_C = M * N;

        total_elements_A += elements_A;
        total_elements_B += elements_B;
        total_elements_C += elements_C;

        stride_A_host.push_back(cutlass::make_cute_packed_stride(StrideA{}, {M, K, 1}));
        stride_B_host.push_back(cutlass::make_cute_packed_stride(StrideB{}, {N, K, 1}));
        stride_C_host.push_back(cutlass::make_cute_packed_stride(StrideC{}, {M, N, 1}));
    }

    block_A.reset(total_elements_A);
    block_B.reset(total_elements_B);
    block_C.reset(total_elements_C);
    block_D.reset(total_elements_C);
    block_ref_D.reset(total_elements_C);
}

/// Initialize operands to be used in the GEMM and reference GEMM
void initialize(const float &alpha,
                const float &beta,
                const std::vector<typename ProblemShape::UnderlyingProblemShape> &problem_sizes_host) {
    const uint32 E = problem_sizes_host.size();

    problem_sizes.reset(E);
    problem_sizes.copy_from_host(problem_sizes_host.data());

    std::vector<ElementA *> ptr_A_host(E);
    std::vector<ElementB *> ptr_B_host(E);
    std::vector<ElementC *> ptr_C_host(E);
    std::vector<ElementC *> ptr_D_host(E);

    for (uint32 i = 0; i < E; ++i) {
        ptr_A_host.at(i) = block_A.get() + offset_A.at(i);
        ptr_B_host.at(i) = block_B.get() + offset_B.at(i);
        ptr_C_host.at(i) = block_C.get() + offset_C.at(i);
        ptr_D_host.at(i) = block_D.get() + offset_C.at(i);
    }

    ptr_A.reset(E);
    ptr_A.copy_from_host(ptr_A_host.data());

    ptr_B.reset(E);
    ptr_B.copy_from_host(ptr_B_host.data());

    ptr_C.reset(E);
    ptr_C.copy_from_host(ptr_C_host.data());

    ptr_D.reset(E);
    ptr_D.copy_from_host(ptr_D_host.data());

    stride_A.reset(E);
    stride_A.copy_from_host(stride_A_host.data());

    stride_B.reset(E);
    stride_B.copy_from_host(stride_B_host.data());

    stride_C.reset(E);
    stride_C.copy_from_host(stride_C_host.data());

    initialize_block(block_A, 2023);
    initialize_block(block_B, 2022);
    initialize_block(block_C, 2021);
}

typename Gemm::Arguments args_from_options(
    const uint32 &E,
    const dim3 &cluster_shape,
    const dim3 &cluster_shape_fallback,
    const float &alpha,
    const float &beta,
    const RasterOrderOptions &raster_order,
    const std::vector<typename ProblemShape::UnderlyingProblemShape> &problem_sizes_host,
    bool host_problem_shapes_available = true) {
    cutlass::KernelHardwareInfo hw_info;
    hipGetDevice(&hw_info.device_id);
    hipDeviceGetAttribute(&hw_info.sm_count, hipDeviceAttributeMultiprocessorCount, hw_info.device_id);

    if (!is_static_v<ClusterShape>) {
        if (size<0>(typename Gemm::GemmKernel::CollectiveMainloop::AtomThrShapeMNK{}) == 2 &&
            (cluster_shape.x < 2 || cluster_shape_fallback.x < 2)) {
            std::cout << "Error: MMA2SMConfig kernel config needs cluster_dim.x >= 2" << std::endl;
        }
        hw_info.cluster_shape = cluster_shape;
        hw_info.cluster_shape_fallback = cluster_shape_fallback;
    }

    typename Gemm::Arguments arguments;
    decltype(arguments.epilogue.thread) fusion_args;
    fusion_args.alpha_ptr = nullptr;
    fusion_args.beta_ptr = nullptr;

    // Single alpha / beta for all groups
    fusion_args.alpha = alpha;
    fusion_args.beta = beta;
    fusion_args.alpha_ptr_array = nullptr;
    fusion_args.beta_ptr_array = nullptr;
    fusion_args.dAlpha = {_0{}, _0{}, 0};
    fusion_args.dBeta = {_0{}, _0{}, 0};

    typename Gemm::GemmKernel::TileSchedulerArguments scheduler;
    scheduler.raster_order = raster_order;

    arguments = typename Gemm::Arguments{cutlass::gemm::GemmUniversalMode::kGrouped,
                                         {static_cast<int>(E),
                                          problem_sizes.get(),
                                          host_problem_shapes_available ? problem_sizes_host.data() : nullptr},
                                         {ptr_A.get(), stride_A.get(), ptr_B.get(), stride_B.get()},
                                         {fusion_args, ptr_C.get(), stride_C.get(), ptr_D.get(), stride_C.get()},
                                         hw_info,
                                         scheduler};

    return arguments;
}

bool verify(const float &alpha,
            const float &beta,
            const std::vector<typename ProblemShape::UnderlyingProblemShape> &problem_sizes_host) {
    const uint32 E = problem_sizes_host.size();

    bool passed = true;
    for (uint32 i = 0; i < E; ++i) {
        auto problem = problem_sizes_host.at(i);
        auto M = get<0>(problem);
        auto N = get<1>(problem);
        auto K = get<2>(problem);
        cutlass::TensorRef ref_A(block_A.get() + offset_A.at(i), Gemm::LayoutA::packed({M, K}));
        cutlass::TensorRef ref_B(block_B.get() + offset_B.at(i), Gemm::LayoutB::packed({K, N}));
        cutlass::TensorRef ref_C(block_C.get() + offset_C.at(i), Gemm::LayoutC::packed({M, N}));
        cutlass::TensorRef ref_D(block_ref_D.get() + offset_C.at(i), Gemm::LayoutD::packed({M, N}));

        // Create instantiation for device reference gemm kernel
        DeviceGemmReference gemm_reference;

        // Launch device reference gemm kernel
        gemm_reference({M, N, K}, alpha, ref_A, ref_B, beta, ref_C, ref_D);

        // Wait for kernel to finish
        hipDeviceSynchronize();

        // Check if output from CUTLASS kernel and reference kernel are equal or not
        passed &= cutlass::reference::device::BlockCompareEqual(
            block_ref_D.get() + offset_C.at(i), block_D.get() + offset_C.at(i), M * N);
    }
    return passed;
}

void grouped_gemm_cuda(const torch::Tensor &A,
                       const torch::Tensor &B,
                       torch::Tensor &output,
                       const torch::Tensor &expert_offsets,
                       const float &alpha,
                       const float &beta) {
    const uint32 E = B.size(0);
    const uint32 N = B.size(1);
    const uint32 K = B.size(2);
    const uint32 TK = A.size(0);

    dim3 cluster_shape = dim3(4, 2, 1);
    dim3 cluster_shape_fallback = dim3(2, 1, 1);
    RasterOrderOptions raster_order = RasterOrderOptions::AlongM;
    std::vector<typename ProblemShape::UnderlyingProblemShape> problem_sizes_host;

    problem_sizes_host.reserve(E);
    for (int i = 0; i < E; i++) {
        const uint32 start = expert_offsets[i].item<int64>();
        const uint32 end = i == E - 1 ? TK : expert_offsets[i + 1].item<int64>();
        const uint32 M = end - start;
        problem_sizes_host.push_back({M, N, K});
    }

    allocate(problem_sizes_host);
    initialize(alpha, beta, problem_sizes_host);

    const bool host_problem_shapes_available = false;

    // Instantiate CUTLASS kernel depending on templates
    Gemm gemm;

    // Create a structure of gemm kernel arguments suitable for invoking an instance of Gemm
    Gemm::Arguments arguments = args_from_options(E,
                                                  cluster_shape,
                                                  cluster_shape_fallback,
                                                  alpha,
                                                  beta,
                                                  raster_order,
                                                  problem_sizes_host,
                                                  host_problem_shapes_available);

    // Using the arguments, query for extra workspace required for matrix multiplication computation
    size_t workspace_size = Gemm::get_workspace_size(arguments);

    // Allocate workspace memory
    cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

    // Check if the problem size is supported or not
    gemm.can_implement(arguments);

    // Initialize CUTLASS kernel with arguments and workspace pointer
    gemm.initialize(arguments, workspace.get());

    // Correctness / Warmup iteration
    gemm.run(/* stream = */ nullptr, /* cuda_adapter = */ nullptr, /* launch_with_pdl = */ false);

    // Check if output from CUTLASS kernel and reference kernel are equal or not
    const bool passed = verify(alpha, beta, problem_sizes_host);

    std::cout << "  Disposition: " << (passed ? "Passed" : "Failed") << std::endl;

    const uint32 iterations = 10;
    if (iterations > 0) {
        GpuTimer timer;
        timer.start();
        for (int iter = 0; iter < iterations; ++iter) {
            gemm.initialize(arguments, workspace.get());
            gemm.run(/* stream = */ nullptr, /* cuda_adapter = */ nullptr, /* launch_with_pdl = */ false);
        }
        timer.stop();

        // Compute average setup and runtime and GFLOPs.
        double gflops = get_gflops(double(timer.elapsed_millis()) / double(iterations) / 1000.0, problem_sizes_host);
        std::cout << "  TFLOPS      : " << gflops / 1000.0 << std::endl;
    }
}
