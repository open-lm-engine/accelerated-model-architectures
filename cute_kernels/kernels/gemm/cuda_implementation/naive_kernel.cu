#include "hip/hip_runtime.h"
// **************************************************
// Copyright (c) 2025, Mayank Mishra
// **************************************************

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include "cute/tensor.hpp"
#include "include/cute_kernels.h"
#include "utils.h"

namespace ck = cute_kernels;
using namespace cute;

using uint32 = ck::uint32;
using fp32 = ck::fp32;

template <typename scalar_t, bool is_A_transposed, bool is_B_transposed>
__global__ void _naive_gemm_cuda_kernel(const scalar_t *_A,
                                        const scalar_t *_B,
                                        const scalar_t *_C,
                                        scalar_t *_output,
                                        const fp32 alpha,
                                        const fp32 beta,
                                        const uint32 M,
                                        const uint32 K,
                                        const uint32 N) {
    const uint32 i = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32 j = blockIdx.x * blockDim.x + threadIdx.x;

    Layout layout_A =
        make_layout(is_A_transposed ? make_shape(K, M) : make_shape(M, K), make_stride(is_A_transposed ? M : K, 1));
    Tensor A = make_tensor(make_gmem_ptr(_A), layout_A);

    Layout layout_B =
        make_layout(is_B_transposed ? make_shape(N, K) : make_shape(K, N), make_stride(is_B_transposed ? K : N, 1));
    Tensor B = make_tensor(make_gmem_ptr(_B), layout_B);

    Layout layout_C = make_layout(make_shape(M, N), make_stride(N, 1));
    Tensor C = make_tensor(make_gmem_ptr(_C), layout_C);
    Tensor output = make_tensor(make_gmem_ptr(_output), layout_C);

    if (i < M && j < N) {
        fp32 accumulator = 0;

        // clang-format off
        #pragma unroll
        // clang-format on
        for (uint32 k = 0; k < K; k++) {
            const scalar_t a = is_A_transposed ? A(k, i) : A(i, k);
            const scalar_t b = is_B_transposed ? B(j, k) : B(k, j);
            accumulator += a * b;
        }

        accumulator *= alpha;

        if (beta != 0) {
            accumulator += beta * C(i, j);
        }

        output(i, j) = accumulator;
    }
}

void naive_gemm_cuda(const torch::Tensor &A,
                     const torch::Tensor &B,
                     std::optional<torch::Tensor> &_C,
                     torch::Tensor &output,
                     const bool &is_A_transposed,
                     const bool &is_B_transposed,
                     const fp32 &alpha,
                     const fp32 &beta,
                     const uint32 &BLOCK_SIZE_M,
                     const uint32 &BLOCK_SIZE_N) {
    CHECK_CUDA_TENSOR(A);
    CHECK_CUDA_TENSOR(B);
    if (_C.has_value()) {
        CHECK_CUDA_TENSOR(_C.value());
    }
    CHECK_CUDA_TENSOR(output);

    const auto [M, N, K] = get_MNK(A, B, is_A_transposed, is_B_transposed);

    CHECK_VALID_THREAD_BLOCK(BLOCK_SIZE_M * BLOCK_SIZE_N);

    dim3 NUM_BLOCKS = dim3(ck::ceil_divide<uint32>(N, BLOCK_SIZE_N), ck::ceil_divide<uint32>(M, BLOCK_SIZE_M), 1);
    dim3 BLOCK_SIZE = dim3(BLOCK_SIZE_N, BLOCK_SIZE_M, 1);

    DISPATCH_FLOAT_KERNEL(A.scalar_type(), "naive_gemm_cuda_kernel", scalar_t, ([&] {
                              const scalar_t *C_data = _C.has_value() ? _C.value().data_ptr<scalar_t>() : nullptr;

                              if (is_A_transposed) {
                                  if (is_B_transposed) {
                                      _naive_gemm_cuda_kernel<scalar_t, true, true>
                                          <<<NUM_BLOCKS, BLOCK_SIZE>>>(A.data_ptr<scalar_t>(),
                                                                       B.data_ptr<scalar_t>(),
                                                                       C_data,
                                                                       output.data_ptr<scalar_t>(),
                                                                       alpha,
                                                                       beta,
                                                                       M,
                                                                       K,
                                                                       N);
                                  } else {
                                      _naive_gemm_cuda_kernel<scalar_t, true, false>
                                          <<<NUM_BLOCKS, BLOCK_SIZE>>>(A.data_ptr<scalar_t>(),
                                                                       B.data_ptr<scalar_t>(),
                                                                       C_data,
                                                                       output.data_ptr<scalar_t>(),
                                                                       alpha,
                                                                       beta,
                                                                       M,
                                                                       K,
                                                                       N);
                                  }
                              } else {
                                  if (is_B_transposed) {
                                      _naive_gemm_cuda_kernel<scalar_t, false, true>
                                          <<<NUM_BLOCKS, BLOCK_SIZE>>>(A.data_ptr<scalar_t>(),
                                                                       B.data_ptr<scalar_t>(),
                                                                       C_data,
                                                                       output.data_ptr<scalar_t>(),
                                                                       alpha,
                                                                       beta,
                                                                       M,
                                                                       K,
                                                                       N);
                                  } else {
                                      _naive_gemm_cuda_kernel<scalar_t, false, false>
                                          <<<NUM_BLOCKS, BLOCK_SIZE>>>(A.data_ptr<scalar_t>(),
                                                                       B.data_ptr<scalar_t>(),
                                                                       C_data,
                                                                       output.data_ptr<scalar_t>(),
                                                                       alpha,
                                                                       beta,
                                                                       M,
                                                                       K,
                                                                       N);
                                  }
                              }
                          }));
}
